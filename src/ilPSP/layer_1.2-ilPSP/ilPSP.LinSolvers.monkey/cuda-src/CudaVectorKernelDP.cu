
#include <hip/hip_runtime.h>
/* 
 * Copyright (C) 2010, Florian Kummer, Technische Universitaet Darmstadt, Fachgebiet fuer Stroemungsmechanik
 *
 * Use, modification and distribution is subject to the Boost Software
 * License, Version 1.0. (See accompanying file LICENSE_1_0.txt or copy at
 * http://www.boost.org/LICENSE_1_0.txt)
 *  
 * Authors: Christoph Busold
 * 
 */
extern "C" __global__ void scale(double* vector, double alpha, unsigned int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < size) {
		vector[idx] *= alpha;
	}
}

extern "C" __global__ void acc(double* x, double* y, double alpha, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < size) {
		x[idx] += y[idx] * alpha;
	}
}

extern "C" __global__ void dnrm2(double* vector, double* result, int size) {
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
	double value;

	sdata[tid] = 0.0;
	if(idx < size) {
		value = vector[idx];
		sdata[tid] += value * value;
	}
	if(idx + blockDim.x < size) {
		value = vector[idx + blockDim.x];
		sdata[tid] += value * value;
	}

	__syncthreads();

	for(int s = blockDim.x / 2; s > 0; s >>= 1) {
		if(tid < s) {
			sdata[tid] += sdata[tid + s];
		}

		__syncthreads();
	}

	if(tid == 0) {
		result[blockIdx.x] = sdata[0];
	}
}

extern "C" __global__ void innerprod(double* x, double* y, double* result, int size) {
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

	sdata[tid] = 0.0;
	if(idx < size) {
		sdata[tid] += x[idx] * y[idx];
	}
	if(idx + blockDim.x < size) {
		sdata[tid] += x[idx + blockDim.x] * y[idx + blockDim.x];
	}

	__syncthreads();

	for(int s = blockDim.x / 2; s > 0; s >>= 1) {
		if(tid < s) {
			sdata[tid] += sdata[tid + s];
		}

		__syncthreads();
	}

	if(tid == 0) {
		result[blockIdx.x] = sdata[0];
	}
}

extern "C" __global__ void mew(double* x, double* y, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < size) {
		x[idx] *= y[idx];
	}
}

extern "C" __global__ void fillSendBuffer(double* sendBuffer, int* indices, double* data, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(idx < size) {
		sendBuffer[idx] = data[indices[idx]];
	}
}
