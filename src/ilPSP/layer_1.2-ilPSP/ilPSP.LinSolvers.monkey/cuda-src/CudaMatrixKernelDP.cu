
#include <hip/hip_runtime.h>
/* 
 * Copyright (C) 2010, Florian Kummer, Technische Universitaet Darmstadt, Fachgebiet fuer Stroemungsmechanik
 *
 * Use, modification and distribution is subject to the Boost Software
 * License, Version 1.0. (See accompanying file LICENSE_1_0.txt or copy at
 * http://www.boost.org/LICENSE_1_0.txt)
 *  
 * Authors: Christoph Busold
 * 
 */

extern __shared__ char smem[];

extern "C" __global__ void sparseMultiply(double* values, int* colIdx, int* rowStart, double* result, double* x, double alpha, double beta, int size) {
	// Dynamically allocated shared memory, should be BlockDim.x + 1 ints (see cuFuncSetSharedSize host code)
	int* sharedRowStart = (int*)smem;
	
	// Indices
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	
	double rowacc = 0.0;
	
	// Each thread loads one element of rowStart
	if(idx < size) {
		sharedRowStart[tid] = rowStart[idx];
	}
	// The first thread loads additionally the next element, needed by the last thread
	if(tid == 0) {
		int loadIdx = min((blockIdx.x + 1) * blockDim.x, size);
		int storIdx = size % blockDim.x > 0 && idx + blockDim.x >= size ? size % blockDim.x : blockDim.x;
		sharedRowStart[storIdx] = rowStart[loadIdx];
	}
	__syncthreads();
	
	if(idx < size) {
		// Multiply and sum up data of this row
		for(int i = sharedRowStart[tid]; i < sharedRowStart[tid + 1]; i++) {
			rowacc += values[i] * x[colIdx[i]];
		}
		
		result[idx] = result[idx] * beta + rowacc * alpha;
	}
}

extern "C" __global__ void accumulateExternal(double* data, int* indices, double* rcvBuffer, double alpha, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(idx < size) {
		data[indices[idx]] += rcvBuffer[idx] * alpha;
	}
}

// In this kernel each block computes multiple cell rows 
// IMPORTANT: All cell rows must have the same number of cells!
//            Otherwise sync in kernel might fail, causing crash or incorrect behaviour!
extern "C" __global__ void blockMultiply2(double* cellData, double* xData, int* cellColIdx, double* result, double alpha, double beta, int cellsize, int cellrowsperblock, int cellsperrow, int stride, int size) {
	// Dynamically allocated shared memory, should be blockDim.x doubles for xData
	double* sharedData = (double*)smem;
	// Start cell index of this thread
	int* start = (int*)&sharedData[blockDim.x];
	// Column of this thread's cell
	int* colIdx = (int*)&start[cellrowsperblock];
	
	// Global index
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	// Number of the cell this thread is in
	int cellid = tid / cellsize;
	// Thread index inside this cell
	int cid = tid % cellsize;
	
	double rowacc = 0.0;
	double value;
	
	// Load start index for every cell row in this block
	if(tid < cellrowsperblock) {
		start[tid] = (cellrowsperblock * blockIdx.x + tid) * cellsperrow;
	}
	__syncthreads();
	
	// Loop over all cells, discard overlapping threads inside because of sync
	for(int i = 0; i < cellsperrow; i++) {
		// Load column index for every cell
		if(tid < cellrowsperblock) {
			colIdx[tid] = cellColIdx[start[tid] + i];
		}
		__syncthreads();
		
		// No overlapping threads
		if(idx < size) {
			// Load x at colIdx location into shared memory
			// colIdx * cellsize is the start index at this column
			// cid is the row index of this thread
			sharedData[tid] = xData[colIdx[cellid] * cellsize + cid];
		}
		__syncthreads();
		
		// No overlapping threads
		if(idx < size) {
			// Loop over all columns of this cell
			for(int col = 0; col < cellsize; col++) {
				// Load value of this column
				// cell * cellsize * cellsize is the start index of the current cell
				// col * cellsize is the start index of the current column
				// cid is the row index of this thread
				value = cellData[(start[cellid] + i) * stride + col * cellsize + cid];
				// Multiply value with x from sharedMemory
				// cellid * cellsize is the offset for the cell this thread is in
				// col is the column index of this loop cycle
				rowacc += value * sharedData[cellid * cellsize + col];
			}
		}
		
		__syncthreads();
	}
	
	// No overlapping threads
	if(idx < size) {
		// Write back result 
		result[idx] = result[idx] * beta + rowacc * alpha;
	}
}

// In this kernel each block computes one cell row (block size equals cell size)
extern "C" __global__ void blockMultiply(double* cellData, double* xData, int* cellColIdx, int* cellRowStart, double* result, double dia, int cellsize, int size) {
	double* sharedData = (double*)smem;
	__shared__ int colIdx;
	__shared__ int start;
	__shared__ int end;
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	
	double rowacc = 0.0;
	double value;
	
	if(tid == 0) {
		start = cellRowStart[blockIdx.x    ];
		end   = cellRowStart[blockIdx.x + 1];
	}
	__syncthreads();
	
	for(int cell = start; cell < end; cell++) {
		if(tid == 0) {
			colIdx = cellColIdx[cell];
		}
		__syncthreads();
		
		if(idx < size) {
			sharedData[tid] = xData[colIdx * cellsize + tid];
		}
		__syncthreads();
		
		if(idx < size) {
			for(int col = 0; col < cellsize; col++) {
				value = cellData[cell * cellsize * cellsize + col * cellsize + tid];
				rowacc += value * sharedData[col];
			}
		}
	}
	
	if(idx < size) {
		rowacc += dia * xData[idx];
		result[idx] += rowacc;
	}
}

// ELLPACKmod format
extern "C" __global__ void ellMultiply(double* valData, int* colIdxData, double* xData, double* result, double alpha, double beta, int size, int colCount, int valStride, int colStride) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// Add offsets to the start of this block's value and column data pointers
	valData    += blockIdx.x * colCount * valStride;
	colIdxData += blockIdx.x * colCount * colStride;
	
	int valIdx;
	int colIdx;
	
	// No sync in this kernel, therefore overlapping threads are discarded here
	if(idx < size) {
		double rowacc = 0.0;
		
		// Loop over all columns
		for(int col = 0; col < colCount; col++) {
			// Index of the value and column index to load
			valIdx = col * valStride + threadIdx.x;
			colIdx = col * colStride + threadIdx.x;
			// Load value and multiply with x at column of this value
			rowacc += valData[valIdx] * xData[colIdxData[colIdx]];
		}
		
		// Write result back
		result[idx] = result[idx] * beta + rowacc * alpha;
	}
}

// ManualCacheELLPACK format
extern "C" __global__ void mcellMultiply(double* valData, unsigned short* colIdxData, int* xSubStart, int* blockSubVector, double* xData, double* result, double alpha, double beta, int size, int colCount, int valStride, int colStride) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	// Add offsets to the start of this block's value and column data pointers
	valData    += blockIdx.x * colCount * valStride;
	colIdxData += blockIdx.x * colCount * colStride;
	
	double* xSub = (double*)smem;
	__shared__ int xStart;
	__shared__ int xLength;
	
	int valIdx;
	unsigned short colIdx;
	
	if(tid == 0) {
		xStart  = xSubStart[blockIdx.x    ];
		xLength = xSubStart[blockIdx.x + 1] - xStart;
	}
	
	__syncthreads();
	
	blockSubVector += xStart;
	int ldIdx = tid;
	
	while(ldIdx < xLength) {
		xSub[ldIdx] = xData[blockSubVector[ldIdx]];
		ldIdx += blockDim.x;
	}
	
	__syncthreads();
	
	// No sync inside this loop, therefore overlapping threads are discarded here
	if(idx < size) {
		double rowacc = 0.0;
		
		// Loop over all columns
		for(int col = 0; col < colCount; col++) {
			// Index of the value and column index to load
			valIdx = col * valStride + tid;
			colIdx = col * colStride + tid;
			// Load value and multiply with x at column of this value
			rowacc += valData[valIdx] * xSub[colIdxData[colIdx]];
		}
		
		// Write result back
		result[idx] = result[idx] * beta + rowacc * alpha;
	}
}
