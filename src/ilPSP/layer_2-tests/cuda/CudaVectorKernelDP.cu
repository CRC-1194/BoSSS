
#include <hip/hip_runtime.h>
extern "C" __global__ void scale(double* vector, double alpha, unsigned int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < size) {
		vector[idx] *= alpha;
	}
}

extern "C" __global__ void acc(double* x, double* y, double alpha, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < size) {
		x[idx] += y[idx] * alpha;
	}
}

extern "C" __global__ void dnrm2(double* vector, double* result, int size) {
	__shared__ double sdata[256];

	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
	double value;

	sdata[tid] = 0.0;
	if(idx < size) {
		value = vector[idx];
		sdata[tid] += value * value;
	}
	if(idx + blockDim.x < size) {
		value = vector[idx + blockDim.x];
		sdata[tid] += value * value;
	}

	__syncthreads();

	for(int s = blockDim.x / 2; s > 0; s >>= 1) {
		if(tid < s) {
			sdata[tid] += sdata[tid + s];
		}

		__syncthreads();
	}

	if(tid == 0) {
		result[blockIdx.x] = sdata[0];
	}
}

extern "C" __global__ void innerprod(double* x, double* y, double* result, int size) {
	__shared__ double sdata[256];

	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

	sdata[tid] = 0.0;
	if(idx < size) {
		sdata[tid] += x[idx] * y[idx];
	}
	if(idx + blockDim.x < size) {
		sdata[tid] += x[idx + blockDim.x] * y[idx + blockDim.x];
	}

	__syncthreads();

	for(int s = blockDim.x / 2; s > 0; s >>= 1) {
		if(tid < s) {
			sdata[tid] += sdata[tid + s];
		}

		__syncthreads();
	}

	if(tid == 0) {
		result[blockIdx.x] = sdata[0];
	}
}
