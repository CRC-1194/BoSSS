
#include <hip/hip_runtime.h>
extern "C" __global__ void sparseMultiply(double* values, int* colIdx, int* rowStart, double* result, double* x, double alpha, double beta, int size) {
	// Dynamically allocated shared memory, should be BlockDim.x + 1 ints (see cuFuncSetSharedSize host code)
	extern __shared__ int sharedRowStart[];
	
	// Indices
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	
	double rowacc = 0.0;
	
	// Each thread loads one element of rowStart
	if(idx < size) {
		sharedRowStart[tid] = rowStart[idx];
	}
	// The first thread loads additionally the next element, needed by the last thread
	if(tid == 0) {
		int lastIdx = min((blockIdx.x + 1) * blockDim.x, size);
		sharedRowStart[blockDim.x] = rowStart[lastIdx];
	}
	__syncthreads();
	
	if(idx < size) {
		// Multiply and sum up data of this row
		for(int i = sharedRowStart[tid]; i < sharedRowStart[tid + 1]; i++) {
			rowacc += values[i] * x[colIdx[i]];
		}

		result[idx] = rowacc * alpha;
	}
}
